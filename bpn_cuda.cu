#include "hip/hip_runtime.h"
#include "bpn_cuda.h"

#define THRES 0.0001 /*Threshold of training error*/

__device__ double computeFunc(double x , Type t){
	if(t == Linear)
		return x;
	
	if(t == Sigmoidal){
		double out = 1 + exp(-x);
		return 1 / out;
	}
}

__device__ double computeDiff(double x , Type t){
	if(t == Linear)
		return 1;
	if(t == Sigmoidal){
		double out = computeFunc(x , t);
		return out * (1 - out);
	}
}

/*CUDA function to feed z value at the input*/
__global__ void forward_propagate_input(double* z_curr
										, double* bias
										, int size
										){

	
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id >= size)
		return;

	z_curr[id] += bias[id];
	return;
}
										

/* CUDA function to propagate z and a values from the input level(level[size - 1]) to the output level(level[0])*/
__global__ void forward_propagate_level(double* a_curr				/*a values of the nodes of current level*/ 
										, double* z_curr			/*z values of the nodes of the current level*/
										, double* weight			/*weight connections between current(l) and previous(l + 1) level*/
										, double* z_prev			/*z values of the nodes of the previous(l + 1) level*/
										, double* bias_curr			/*bias value of the current level*/
										, int prev					/*size of previous level*/
										, int curr					/*size of current level*/
										, Type t					/*threshold function type of the current level*/
										){

	int id = threadIdx.x + blockDim.x * blockIdx.x;

	if(id >= curr)
		return;

	int i;

	a_curr[id] = 0;
	for(i = 0 ; i < prev ; i ++)
		a_curr[id] += weight[id * prev + i] * z_prev[i];

	a_curr[id] += bias_curr[id];
	z_curr[id] = computeFunc(a_curr[id] , t);

	return;

}



/*Function to initialize the delta-values at the output*/

__global__ void reverse_propagate_output(double *delta_curr					/*delta value of the current level*/ 
										 , double* z_curr					/*z value of the current level*/
										 , double* target					/*target value at the output*/
										 , double* a_curr					/*a values at the output*/
										 , int size							/*size of the output level*/
										 , Type t							/*threshold value at the output*/
										 ){
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if(id >= size)
		return;
	
	delta_curr[id] = (z_curr[id] - target[id]) * computeDiff(a_curr[id] , t);

	return;
}

/*Function to propagate the delta-values from one level to another*/
__global__ void reverse_propagate_level(double *delta_curr				
										, double *delta_next
										, double* weight_next
										, double* a_curr
										, Type t						/*Type of threshold function at the current level*/
										, int size_next					/*Number of nodes at the next level*/ 
										, int size						/*Number of nodes at the current level*/
										){

	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if(id >= size)
		return;

	int i;
	delta_curr[id] = 0;

	for(i = 0 ; i < size_next ; i ++)
		delta_curr[id] += delta_next[i] * weight_next[i * size + id];

	delta_curr[id] *= computeDiff(a_curr[id] , t);

	return;
}
/*Function that updates weight between two levels*/
__global__ void weight_update(double* weight						/*Weight between current and previous level*/
							  , double* delta_curr
							  , double* z_prev
							  , int size							/*Size of current level*/
							  , int size_prev
							  , double rate							/*Rate at which the weights are updated*/
							  ){
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if(id >= size * size_prev)
		return;

	int curr_node = id / size_prev;
	int prev_node = id % size_prev;

	weight[curr_node * size_prev + prev_node] -= rate * delta_curr[curr_node] * z_prev[prev_node];
	return;
}

/*Function that updates the bias of a level*/
__global__ void bias_update(double* delta_curr
							, double* bias                          /*Bias of the current level*/
							, int size
							, double rate){
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	if(id >= size)
		return;

	bias[id] -= rate * delta_curr[id];

	return;
}


void copyBPNinput(BPN_CUDA* network , double *input){
	int inputLevel = network->noLevels - 1;
	hipMemcpy(network->z_val + network->noNodes - network->nodeSize[inputLevel] , input , network->nodeSize[network->noLevels - 1] * sizeof(double) , hipMemcpyHostToDevice);
	hipMemcpy(network->a_val + network->noNodes - network->nodeSize[inputLevel], input , network->nodeSize[network->noLevels - 1] * sizeof(double) , hipMemcpyHostToDevice);
	return;
}

/*Function to compute the forward propagation of values*/
void forward(BPN_CUDA* network , double *input){
	int i , sizePrev , sizeCurr;
	double* a_curr , double* z_curr , double* weight , double* z_prev , double* bias_curr;
	Type t = network->type[network->noLevels - 2];

	copyBPNinput(network , input);



	sizePrev = network->nodeSize[network->noLevels - 1];
	sizeCurr = network->nodeSize[network->noLevels - 2];
	a_curr = network->a_val + network->noNodes - sizePrev - sizeCurr;
	z_curr = network->z_val + network->noNodes - sizePrev - sizeCurr;
	weight = network->weight + network->noWeight - sizePrev * sizeCurr;
	z_prev = network->z_val + network->noNodes - sizePrev;
	bias_curr = network->bias + network->noNodes - sizePrev - sizeCurr;

	forward_propagate_input<<<(sizePrev / 1024 + 1) , (sizePrev > 1024 ? 1024 : sizePrev)>>>(z_prev , bias_curr + sizeCurr , sizePrev);

	for(i = network->noLevels - 2 ; i > -1 ; i --){
		forward_propagate_level<<<(sizeCurr / 1024 + 1) , (sizeCurr > 1024 ? 1024 : sizeCurr)>>>(a_curr , z_curr , weight , z_prev , bias_curr , sizePrev , sizeCurr , t);
		
		if(i == 0)
			break;

		
		if(i == 0)
			break;

		z_prev = z_curr;

		sizePrev = sizeCurr;
		sizeCurr = network->nodeSize[i - 1];

		a_curr = a_curr - sizeCurr;
		z_curr = z_curr - sizeCurr;
		bias_curr = bias_curr - sizeCurr;
		weight = weight - sizePrev * sizeCurr;

	}


}
/*Function to compute the reverse propagation of values*/
double reverse(BPN_CUDA* network , double* target){

	double *delta_curr , *z_curr , *a_curr , *delta_next , *weight_next , *z_curr_h , *target_d;
	delta_curr = network->delta;
	z_curr = network->z_val;
	a_curr = network->a_val;
	int size = network->nodeSize[0] , size_next;
	Type t = network->type[0];
	double error = 0;

	z_curr_h = new double[size];
	hipMemcpy(z_curr_h , z_curr , size * sizeof(double) , hipMemcpyDeviceToHost);

	hipMalloc((void**)&target_d , size * sizeof(double));
	hipMemcpy(target_d , target , size * sizeof(double) , hipMemcpyHostToDevice);
	
	for(int i = 0 ; i < network->nodeSize[0] ; i ++)
		error += (target[i] - z_curr_h[i]) * (target[i] - z_curr_h[i]);

	reverse_propagate_output<<<(size / 1024 + 1) , (size > 1024 ? 1024 : size)>>>(delta_curr , z_curr , target_d , a_curr , size , t);

	delta_next = delta_curr;
	weight_next = network->weight;

	z_curr = NULL;
	delta_curr = delta_curr + size;
	a_curr = a_curr + size;
	t = network->type[1];
	size_next = size;
	size = network->nodeSize[1];


	for(int i = 1 ; i < network->noLevels ; i ++){
		reverse_propagate_level<<<(size / 1024 + 1) , (size > 1024 ? 1024 : size)>>>(delta_curr , delta_next , weight_next , a_curr , t , size_next , size);

		if(i == network->noLevels - 1)
			break;

		delta_next = delta_curr;
		weight_next = weight_next + size * size_next;

		a_curr = a_curr + size;
		delta_curr = delta_curr + size;

		size_next = size;
		size = network->nodeSize[i + 1];
		t = network->type[i + 1];
		
	}

	return error;

}


void weight_bias_update(BPN_CUDA* network , double rate){

	double* weight = network->weight;
	double* delta_curr = network->delta;

	int size = network->nodeSize[0];

	double* z_prev = network->z_val + size;
	double* bias = network->bias;

	int size_prev = network->nodeSize[1];
	for(int i = 1 ; i < network->noLevels ; i ++){
		weight_update<<<(size * size_prev / 1024 + 1) , (size * size_prev > 1024 ? 1024 : size * size_prev)>>>(weight , delta_curr , z_prev , size , size_prev , rate);
		bias_update<<<(size / 1024 + 1) , (size > 1024 ? 1024 : size)>>>(delta_curr , bias , size , rate);

		if(i == network->noLevels - 1)
			break;

		weight = weight + size * size_prev;
		delta_curr = delta_curr + size;
		z_prev = z_prev + size_prev;
		bias = bias + size;

		size = size_prev;
		size_prev = network->nodeSize[i + 1];
	}
}

int train(BPN_CUDA* network , double* input , double* output , int dataset_no , int input_size , int output_size){
	double error;
	double *ip , *op;
	int count = 0;
	while(true){
		error = 0;
		ip = input;
		op = output;
		for(int i = 0 ; i < dataset_no ; i ++){
			forward(network , ip);
			error += reverse(network , op);
			weight_bias_update(network , network->training_rate);
			ip = ip + input_size;
			op = op + output_size;
		}

		//printf("%f\n" , error);

		if(error < THRES || count == 1000)
			break;

		count ++;
	}
	return count;
}

void initialize(BPN_CUDA* network , int* noNodes , int levels , Type* type , double rate){
	
	network->noLevels = levels;
	network->nodeSize = new int[levels];
	network->type = new Type[levels];
	network->training_rate = rate;

	for(int i = 0 ; i < levels ; i ++){
		network->nodeSize[i] = noNodes[i];
		network->type[i] = type[i];
	}

	int numNodes = 0;
	int numWeights = 0;
	for(int i = 0 ; i < levels ; i ++){
		numNodes += noNodes[i];

		if(i == 0)
			continue;

		numWeights += noNodes[i] * noNodes[i - 1];
	}
	
	double *device_mem;
	
	hipMalloc((void**)&network->a_val , numNodes * sizeof(double));
	hipMalloc((void**)&network->z_val , numNodes * sizeof(double));
	hipMalloc((void**)&network->delta , numNodes * sizeof(double));
	hipMalloc((void**)&network->bias , numNodes * sizeof(double));
	hipMalloc((void**)&network->weight , numWeights * sizeof(double));

	network->noNodes = numNodes;
	network->noWeight = numWeights;

	double* initval = new double[numNodes];
	double* initweight = new double[numWeights];

	time_t t;
	srand((unsigned)time(&t));

	for(int i = 0 ; i < numWeights ; i ++){
		
		initweight[i] = (double)(rand() % 50) / 100000.0;
		initweight[i] = initweight[i] == 0.0 ? 0.0001 : initweight[i];

		if(i < numNodes){
			initval[i] = (double)(rand() % 50) / 100000.0;
			initval[i] = initval[i] == 0.0 ? 0.0001 : initval[i];
		}

	}

	if(numWeights == 2)//If number of weight connections is true, then no-weights = no-nodes + 1
		initval[2] = 0.0001;

	hipMemcpy(network->a_val , initval , numNodes * sizeof(double) , hipMemcpyHostToDevice);	
	hipMemcpy(network->z_val , initval , numNodes * sizeof(double) , hipMemcpyHostToDevice);	
	hipMemcpy(network->delta , initval , numNodes * sizeof(double) , hipMemcpyHostToDevice);	
	hipMemcpy(network->bias , initval , numNodes * sizeof(double) , hipMemcpyHostToDevice);	
	hipMemcpy(network->weight , initweight , numWeights * sizeof(double) , hipMemcpyHostToDevice);

}

void returnOutput(BPN_CUDA* network , double* input , double* output){
	int size = network->nodeSize[0];
	forward(network , input);

	hipMemcpy(output , network->z_val , size * sizeof(double) , hipMemcpyDeviceToHost);

	return;
}